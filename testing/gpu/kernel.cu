#include "hip/hip_runtime.h"
#include <iostream>
#include "cmf.h"
#include "box.h"

using cmf::print;

__global__ void K_FillBlock(cmf::BlockArray<double, 1> arLb, box bx)
{
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int j = blockIdx.y*blockDim.y+threadIdx.y;
    int k = blockIdx.z*blockDim.z+threadIdx.z;
    if (i>=arLb.imin && i<arLb.imax && j>=arLb.jmin && j<arLb.jmax && k>=arLb.kmin && k<arLb.kmax)
    {
        arLb(0, i, j, k) = i;
        arLb(1, i, j, k) = j;
        arLb(2, i, j, k) = k;
        
        arLb(3, i, j, k) = bx.xmin + bx.dx*((double)i+0.5);
        arLb(4, i, j, k) = bx.ymin + bx.dy*((double)j+0.5);
        arLb(5, i, j, k) = bx.zmin + bx.dz*((double)k+0.5);
    }
}

void FillBlockGpu(cmf::BlockArray<double, 1>& arLb, const box& bx)
{
    int nx = arLb.imax-arLb.imin;
    int ny = arLb.jmax-arLb.jmin;
    int nz = arLb.kmax-arLb.kmin;
    dim3 block(4,4,4);
    dim3 grid (0,0,0);
    grid.x = (nx + block.x - 1)/block.x;
    grid.y = (ny + block.y - 1)/block.y;
    grid.z = (nz + block.z - 1)/block.z;
    K_FillBlock<<<grid, block>>>(arLb, bx);
}